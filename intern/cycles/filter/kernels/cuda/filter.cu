
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2017 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

#ifdef __CUDA_ARCH__

#include "util_cuda_capabilities.h"

#include "../../filter_compat_cuda.h"

#include "../../filter_kernel.h"

/* kernels */

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_shadow(int sample,
                                 TilesInfo *tiles,
                                 float *unfilteredA,
                                 float *unfilteredB,
                                 float *sampleVariance,
                                 float *sampleVarianceV,
                                 float *bufferVariance,
                                 int4 prefilter_rect,
                                 int buffer_pass_stride,
                                 int buffer_denoising_offset,
                                 bool use_gradients,
                                 bool use_split_variance)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_divide_shadow(sample,
		                            tiles,
		                            x, y,
		                            unfilteredA,
		                            unfilteredB,
		                            sampleVariance,
		                            sampleVarianceV,
		                            bufferVariance,
		                            prefilter_rect,
		                            buffer_pass_stride,
		                            buffer_denoising_offset,
		                            use_gradients,
		                            use_split_variance);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_get_feature(int sample,
                               TilesInfo *tiles,
                               int m_offset,
                               int v_offset,
                               float *mean,
                               float *variance,
                               int4 prefilter_rect,
                               int buffer_pass_stride,
                               int buffer_denoising_offset,
                               bool use_cross_denoising,
                               bool use_split_variance)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_get_feature(sample,
		                          tiles,
		                          m_offset, v_offset,
		                          x, y,
		                          mean, variance,
		                          prefilter_rect,
		                          buffer_pass_stride,
		                          buffer_denoising_offset,
		                          use_cross_denoising,
		                          use_split_variance);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_combine_halves(float *mean, float *variance, float *a, float *b, int4 prefilter_rect, int r)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_combine_halves(x, y, mean, variance, a, b, prefilter_rect, r);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_construct_transform(int sample, float const* __restrict__ buffer,
                                       float *transform, int *rank,
                                       int4 filter_area, int4 rect,
                                       int half_window, float pca_threshold)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int *l_rank = rank + y*filter_area.z + x;
		float *l_transform = transform + y*filter_area.z + x;
		kernel_filter_construct_transform(sample, buffer,
		                                  x + filter_area.x, y + filter_area.y,
		                                  rect, l_transform, l_rank,
		                                  half_window, pca_threshold,
		                                  filter_area.z*filter_area.w,
		                                  threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_combined(float *buffers, int sample,
                                   int offset, int stride,
                                   int4 filter_area, int pass_stride,
                                   int no_denoising_offset)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		kernel_filter_divide_combined(x + filter_area.x, y + filter_area.y, sample, buffers, offset, stride, pass_stride, no_denoising_offset);
	}
}


extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_difference(int dx, int dy,
                                       float ccl_readonly_ptr weightImage,
                                       float ccl_readonly_ptr varianceImage,
                                       float *differenceImage,
                                       int4 rect, int w,
                                       int channel_offset,
                                       float a, float k_2) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_calc_difference(x, y, dx, dy, weightImage, varianceImage, differenceImage, rect, w, channel_offset, a, k_2);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_blur(float ccl_readonly_ptr differenceImage, float *outImage, int4 rect, int w, int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_blur(x, y, differenceImage, outImage, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_calc_weight(float ccl_readonly_ptr differenceImage, float *outImage, int4 rect, int w, int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_calc_weight(x, y, differenceImage, outImage, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_update_output(int dx, int dy,
                                     float ccl_readonly_ptr differenceImage,
                                     float ccl_readonly_ptr image,
                                     float *outImage, float *accumImage,
                                     int4 rect, int w,
                                     int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_update_output(x, y, dx, dy, differenceImage, image, outImage, accumImage, rect, w, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_normalize(float *outImage, float ccl_readonly_ptr accumImage, int4 rect, int w) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + rect.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y + rect.y;
	if(x < rect.z && y < rect.w) {
		kernel_filter_nlm_normalize(x, y, outImage, accumImage, rect, w);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_nlm_construct_gramian(int dx, int dy,
                                         float ccl_readonly_ptr differenceImage,
                                         float ccl_readonly_ptr buffer,
                                         float *color_pass,
                                         float *variance_pass,
                                         float const* __restrict__ transform,
                                         int *rank,
                                         float *XtWX,
                                         float3 *XtWY,
                                         int4 rect,
                                         int4 filter_rect,
                                         int w, int h, int f) {
	int x = blockDim.x*blockIdx.x + threadIdx.x + max(0, rect.x-filter_rect.x);
	int y = blockDim.y*blockIdx.y + threadIdx.y + max(0, rect.y-filter_rect.y);
	if(x < min(filter_rect.z, rect.z-filter_rect.x) && y < min(filter_rect.w, rect.w-filter_rect.y)) {
		kernel_filter_nlm_construct_gramian(x, y, dx, dy, differenceImage, buffer, color_pass, variance_pass, transform, rank, XtWX, XtWY, rect, filter_rect, w, h, f);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_finalize(int w, int h,
                            float *buffer, int *rank,
                            float *XtWX, float3 *XtWY,
                            int4 filter_area, int4 buffer_params,
                            int sample) {
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		int storage_ofs = y*filter_area.z+x;
		rank += storage_ofs;
		XtWX += storage_ofs;
		XtWY += storage_ofs;
		kernel_filter_finalize(x, y, w, h, buffer, rank, filter_area.z*filter_area.w, XtWX, XtWY, buffer_params, sample);
	}
}

#endif

